

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <math.h>
#include <stdio.h>


__global__ void iteration_kernel(int* counts, hiprandState* states, const int n) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n) {
        double x,y;
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);
        if(sqrt((x*x)+(y*y)) <= 1.0) counts[tid] += 1;
    }
}

__global__ void rand_init_kernel(hiprandState *states, int n) {
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n) hiprand_init(tid, tid, 0, &states[tid]);
}

int main(int argc, char* argv[]) {
    
    int NUM_ITERATIONS = 100000000;
    int NUM_THREADS = 10000;
    int BLOCK_SIZE = 1024;

    hiprandState *dev_random;
    hipMalloc((void**)&dev_random, NUM_THREADS*sizeof(hiprandState));
    
    dim3 grid_dim((NUM_THREADS+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    dim3 block_dim(BLOCK_SIZE, 1, 1);

    rand_init_kernel<<<grid_dim,block_dim>>>(dev_random, NUM_THREADS);

    int* dev_counts;
    hipMalloc((void**)&dev_counts, NUM_THREADS*sizeof(int));
    hipMemset(dev_counts, 0, NUM_THREADS*sizeof(int));

    int NUM_ITS_PER_THREAD = NUM_ITERATIONS / NUM_THREADS;
    for(int it = 0; it < NUM_ITS_PER_THREAD; it++) {
        iteration_kernel<<<grid_dim,block_dim>>>(dev_counts, dev_random, NUM_THREADS);   
    }
    hipDeviceSynchronize();
    int* res = (int*)malloc(sizeof(int)*NUM_THREADS);
    hipMemcpy(res, dev_counts, sizeof(int)*NUM_THREADS, hipMemcpyDeviceToHost);    
    hipFree(dev_counts);
    hipFree(dev_random);

    int count = 0;
    for(int i = 0; i < NUM_THREADS; i++) {
        count += res[i];
    }

    printf("PI = %f\n", 4*(double)count/(double)NUM_ITERATIONS);
    return 0;
}